#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
using namespace std;
#define TILE_WIDTH 32

#define WIDTH 1080
#define HEIGHT 1080 

#define MaxRGB 256 

typedef struct {
	unsigned int red;
	unsigned int green;
	unsigned int blue;
} RGB;

typedef struct {
	RGB* image;
	unsigned int width;
	unsigned int height;
} Mandelbrot;

__global__ void mandelbrotKernel(Mandelbrot mandelbrot, double* cr, double* ci) {
	
	__shared__ double ci_s[HEIGHT];
	__shared__ double cr_s[WIDTH];
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (row > mandelbrot.height || col > mandelbrot.width) return;

	int index = row * mandelbrot.width + col;

	ci_s[row] = ci[row];
	cr_s[col] = cr[col];

	int i = 0;
	double zr = 0.0;
	double zi = 0.0;

	const int maxIterations = 500; 

	while (i < maxIterations && zr * zr + zi * zi < 4.0) {// converge catre infinit?
		double fz = zr * zr - zi * zi + cr_s[col];
		zi = 2.0 * zr * zi + ci_s[row];
		zr = fz;
		i++;
	}

	int r, g, b;
	int maxRGB = 256;
	int max3 = maxRGB;
	double t = (double)i / (double)maxIterations;
	i = (int)(t * (double)max3);
	b = i / (maxRGB * maxRGB);
	int nn = i - b * maxRGB;
	r = nn / maxRGB;
	g = nn - r * maxRGB;
	mandelbrot.image[index].red = r;
	mandelbrot.image[index].green = g;
	mandelbrot.image[index].blue = b;
}

int getCValues(double* c, int state, double beginRange, double endRange, double minVal, double maxVal);
hipError_t mandelbrotSetWithCUDA(Mandelbrot mandelbrot, double* cr, double* ci);

int main()
{
	unsigned int width,height,maxRGB;

	width = WIDTH; 
	height = HEIGHT; 
	maxRGB = MaxRGB; 
	Mandelbrot mandelbrot;

	double* cr;
	double* ci;

	double minValR = -2.0;
	double maxValR = 1.0;
	double minValI = -1.5;
	double maxValI = 1.5;

	size_t size;

	mandelbrot.width = width;
	mandelbrot.height = height;
	
	size = width * height * sizeof(RGB);
	mandelbrot.image = (RGB*)malloc(size);


	size = width * sizeof(double);
	cr = (double*)malloc(size);
	size = height * sizeof(double);
	ci = (double*)malloc(size);

	getCValues(cr, 0, 0, width, minValR, maxValR);
	getCValues(ci, 0, 0, height, minValI, maxValI);

	hipError_t cudaStatus = mandelbrotSetWithCUDA(mandelbrot, cr, ci);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Kernel fail");
	}
	else {
		printf("Generating image...\n");
		ofstream fout("output_image.ppm");
		fout << "P3" << endl;
		fout << mandelbrot.width << " " << mandelbrot.height << endl;
		fout << maxRGB << endl;
		
		for (int h = 0; h < height; h++) {
			
			for (int w = 0; w < width; w += 2) {			
				int index = h * width + w;
				fout << mandelbrot.image[index].red << " " << mandelbrot.image[index].green << " " << mandelbrot.image[index].blue << " ";
				fout << mandelbrot.image[index + 1].red<< " " << mandelbrot.image[index + 1].green << " " << mandelbrot.image[index + 1].blue << " ";
			}
			fout << endl;
		}
		fout.close();

		printf("Succes!\n");
	}
	return 0;
}

int getCValues(double* c, int state, double beginRange, double endRange, double minVal, double maxVal) {
	if (state < endRange) {
		c[state] = ((state - beginRange) / (endRange - beginRange))*(maxVal - minVal) + minVal;
		return getCValues(c, state + 1, beginRange, endRange, minVal, maxVal);
	}
	else {
		return 0;
	}
}

hipError_t mandelbrotSetWithCUDA(Mandelbrot mandelbrot, double* cr, double* ci)
{
	hipError_t cudaStatus;

	unsigned int width = mandelbrot.width;
	unsigned int height = mandelbrot.height;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice fail");
	}

	Mandelbrot mandelbrot_d;
	mandelbrot_d.width = width;
	mandelbrot_d.height = height;
	size_t  mandlebortSize = width * height * sizeof(RGB);
	cudaStatus = hipMalloc((void **)&mandelbrot_d.image, mandlebortSize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc fail");
	}

	double* cr_d;
	size_t CRealSize = width * sizeof(double);
	cudaStatus = hipMalloc((void**)&cr_d, CRealSize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Real hipMalloc fail");
	}

	double* ci_d;
	size_t  CImagSize = height * sizeof(double);
	cudaStatus = hipMalloc((void**)&ci_d, CImagSize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Imaginari hipMalloc fail");
	}

	cudaStatus = hipMemcpy(cr_d, cr, CRealSize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Real hipMemcpy fail");
	}

	cudaStatus = hipMemcpy(ci_d, ci, CImagSize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Imaginari hipMemcpy fail");
	}

	int blocks_x = (width + TILE_WIDTH - 1) / TILE_WIDTH;
	int blocks_y = (height + TILE_WIDTH - 1) / TILE_WIDTH;

	dim3 dimGrid(blocks_x, blocks_y, 1);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

	mandelbrotKernel << <dimGrid, dimBlock >> > (mandelbrot_d, cr_d, ci_d);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "mandelbrotKernel fail: %s\n", hipGetErrorString(cudaStatus));
	}


	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize fail : %d\n", cudaStatus);
		goto DeallocateMemory;
	}

	cudaStatus = hipMemcpy(mandelbrot.image, mandelbrot_d.image, mandlebortSize, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy fail");
	}

DeallocateMemory:
	hipFree(mandelbrot_d.image);
	hipFree(cr_d);
	hipFree(ci_d);
	return cudaStatus;
}
